#include "hip/hip_runtime.h"
#include "Variable.hh"
#include "FitManager.hh"
#include "UnbinnedDataSet.hh"
#include "BinnedDataSet.hh"
#include "ThreeBodiesPdf.hh"
#include "VoigtianPdf.hh"
#include "VoigtianThreshPdf.hh"
#include "FitControl.hh"
#include "TDatime.h"
#include "TH1F.h"
#include "TH2F.h"
#include "TMath.h"
#include "TStyle.h"
#include "TCanvas.h"
#include "TMinuit.hh"
#include "TPaveText.h"
#include "TString.h"
#include "AddPdf.hh"
#include "TRandom.hh"
#include "TRandom3.h"
#include "TLatex.h"
#include "TMultiGraph.h"
#include "TAxis.h"
#include "TColor.h"
#include "TFile.h"
#include "TGraph.h"
#include "TLegend.h"
#include "TPaveStats.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <sys/times.h>
#include <sys/time.h>
#include <fstream>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

//#define SCATTERPLOTS 1
#define GOODPLOTS 30
//#define READING 1
#define NLLBEST 1
//#define NEGSIG 0.25
//#define GAMMAFIX 0.03
//#define MEANFIX 1051.4 
//#define TOYWRITE 1
//#define STARTINGPOINTS 1

using namespace std;

timeval startTime, stopTime, totalTime;
timeval startTimeRead, stopTimeRead, totalTimeRead;
clock_t startCPU, stopCPU;
clock_t startCPURead, stopCPURead;
tms startProc, stopProc;
tms startProcRead, stopProcRead;

////////////////////////////////////////////////////////////////////////////////////
  //BACKGROUND FUNCTION//

Double_t fondo (Double_t x) {

    Double_t ret = (x<1)||(x>2)? 0:(sqrt( pow(x+3.0967,4) + pow(3.0967,4) + pow(1.01946,4) - 2*pow(x+3.0967,2)*pow(3.0967,2) - 2*pow(3.0967,2)*pow(1.01946,2) - 2*pow(x+3.0967,2)*pow(1.01946,2) ) * sqrt( pow(5.279,4) + pow(x+3.0967,4) + pow(0.493677,4) - 2*pow(5.279,2)*pow(x+3.0967,2) - 2*pow(5.279,2)*pow(0.493677,2) - 2*pow(x+3.0967,2)*pow(0.493677,2) ) / (x+3.0967));

    return ret;
}

////////////////////////////////////////////////////////////////////////////////////
  //ROUND FUNCTION//

Double_t approximate (Double_t x) {

    Double_t result;
	
	if(x<=0) result = ((Double_t)floor(x*100000+0.5))/100000.0;
	 else return x;
	 
    return result;
}

////////////////////////////////////////////////////////////////////////////////////
  //CHISQUARE CALCULATOR//

Double_t chisquare(TH1F& dataHist,TH1F& pdfHist,Variable *xvar){

Double_t chi=0;

//#pragma omp parallel for
for(int y=0;y<=xvar->numbins;y++){

    Double_t data;
    Double_t pdf;
    Double_t termine=0;

   // pdf=floor(pdfHist.GetBinContent(y+1)+0.5); //ROUND TO INTEGER
   // data=floor(dataHist.GetBinContent(y+1)+0.5);
    pdf=pdfHist.GetBinContent(y+1); //ROUND TO INTEGER
    data=dataHist.GetBinContent(y+1);    
	
    termine=pow(pdf-data,2);

    if(pdf<1){
        chi+=termine;
    }else{
    termine/=pdf;
    chi+=termine;
    }
    //cout<<"Termine="<<termine<<" Data ="<<dataHist.GetBinContent(y+1)<<" PDF = "<<pdf<<" CHI = "<<chi<<endl;
}

return chi;

}

////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {
    //hipSetDevice(0);
    hipFree(0);
    hipError_t a;
    hipCtx_t pctx;
    hipDevice_t device;
/*    hipInit(0);
    hipDeviceGet(&device, 0);
    std::cout << "DeviceGet : " << a << std::endl;
    //hipCtxCreate(&pctx, hipDeviceScheduleAuto, device ); // explicit context here
    std::cout << "CtxCreate : " << a << std::endl;
    assert(a == hipSuccess);
    a = hipCtxPopCurrent(&pctx);
    std::cout << "hipCtxPopCurrent : " << a << std::endl;
    //cuCtxDetach(pctx);
    //cuCtxDetach(pctx);
    //a = hipCtxPopCurrent(&pctx);
    //assert(a == hipSuccess);
    std::cout << "Initialized CUDA" << std::endl;
 
   //hipInit();
    hipDeviceReset();
   // hipSetDevice(0);
    gStyle->SetOptStat(1111111111);
  */  gStyle->SetFillColor(0);
////////////////////////////////////////////////////////////////////////////////////
//INPUTS
////////////////////////////////////////////////////////////////////////////////////
  //Events & toys
  int events=2500;//2543;//5000;//
  int iter = atoi(argv[1]);
  
  //Randomizer MultiProcesses
  int rndInt = 1;
  #ifndef READING
  if(argc>2) rndInt = atoi(argv[2]);
  else rndInt = 5;
  #endif
  
  //Reader MultiProcesses
  int toys = 0;
  #ifdef READING
  if(argc>2) toys = atoi(argv[2]); //Dividing Toys Among MultiProcesses  
  #endif
  #ifndef READING
  if(argc>3) toys = atoi(argv[3]); 
  #endif
////////////////////////////////////////////////////////////////////////////////////

  TString name = "";
  switch (iter) {
  case 100: name = "100";
    break;
  case 1000: name = "1k";
    break;
  case 10000: name = "10k";
    break;
  case 100000: name = "100k";
    break;
  case 500000: name = "500k";
    break;
  case 1000000: name = "1M";
    break;
  case 5000000: name = "5M";
    break;
  case 10000000: name = "10M";
    break;
  case 50000000: name = "50M";
    break;
  case 100000000: name = "100M";
    break;

  default: name = argv[1];
    break;
  }


  cout<<endl;
  cout<<"==================================================================================================="<<endl;
  cout<<"=========================== Starting Toy MC : "<<name<<" iterations ==========================="<<endl;
  cout<<"==================================================================================================="<<endl;
  cout<<endl;

////////////////////////////////////////////////////////////////////////////////////
  //MAIN VARIABLE//
  Variable* xvar = new Variable("Mass",1.02,0.001,1.008, 1.568);
  xvar->numbins = 28;

////////////////////////////////////////////////////////////////////////////////////
  //NAMES
  char canvasname[256];
  char bufferstring[256];
  char filename[256];
  char pdfname[256];
  char histoname[256];
  char dircname[256];
  int gFix=0; 
  int mFix=0; 
  int sNeg=0;
  
////////////////////////////////////////////////////////////////////////////////////
  //PHYSICAL PARAMETERS//
  float MassStart = 1.0467 - 0.003*3.0;
  float MassEnd = 1.0467 + 0.003*3.0;
  float myWidthStart =  0.005;//0.017; //0.01
  float myWidthEnd = 0.0605; //0.08
  float GammaStart = 0.01; //0.0017;
  float GammaEnd = 0.0653;

  #ifdef SCATTERPLOTS
////////////////////////////////////////////////////////////////////////////////////
   //GRID FOR SCATTERPLOTS//
////////////////////////////////////////////////////////////////////////////////////
   TMultiGraph *grid = new TMultiGraph();

////////////////////////////////////////////////////////////////////////////////////
 //STARTING POINTS GRAPHS//
////////////////////////////////////////////////////////////////////////////////////
    //Double_t Width1[1]={0.005};
	 Double_t Width1[1]={0.015};
    Double_t Width2[1]={0.0235};
    Double_t Width3[1]={0.0420};
    Double_t Width4[1]={0.0605};

    Double_t Massa1[1]={1.0407};
    Double_t Massa2[1]={1.0507};

    TGraph* Start1 = new TGraph(1,Massa1,Width1);
    Start1->SetMarkerStyle(29);
    Start1->SetMarkerColor(1);
    Start1->SetMarkerSize(1.3);

    TGraph* Start2 = new TGraph(1,Massa2,Width1);
    Start2->SetMarkerStyle(29);
    Start2->SetMarkerColor(2);
    Start2->SetMarkerSize(1.3);
   
    TGraph* Start3 = new TGraph(1,Massa1,Width2);
    Start3->SetMarkerStyle(29);
    Start3->SetMarkerColor(3);
    Start3->SetMarkerSize(1.3);

    TGraph* Start4 = new TGraph(1,Massa2,Width2);
    Start4->SetMarkerStyle(29);
    Start4->SetMarkerColor(4);
    Start4->SetMarkerSize(1.3);

    TGraph* Start5 = new TGraph(1,Massa1,Width3);
    Start5->SetMarkerStyle(29);
    Start5->SetMarkerColor(9);
    Start5->SetMarkerSize(1.3);
    
    TGraph* Start6 = new TGraph(1,Massa2,Width3);
    Start6->SetMarkerStyle(29);
    Start6->SetMarkerColor(6);
    Start6->SetMarkerSize(1.3);

    TGraph* Start7 = new TGraph(1,Massa1,Width4);
    Start7->SetMarkerStyle(29);
    Start7->SetMarkerColor(7);
    Start7->SetMarkerSize(1.3);
    
    TGraph* Start8 = new TGraph(1,Massa2,Width4);
    Start8->SetMarkerStyle(29);
    Start8->SetMarkerColor(8);
    Start8->SetMarkerSize(1.3);

////////////////////////////////////////////////////////////////////////////////////
//AXES FOR GRID (BINNING)
////////////////////////////////////////////////////////////////////////////////////

	 Double_t y1[2] = {0.02,0.02};
    Double_t y2[2] = {0.04,0.04};
    Double_t y3[2] = {0.06,0.06};

    Double_t x1[2]={1.048,1.048};

    Double_t MeanLim[2]={MassStart,MassEnd};
    Double_t MeanMax[2]={MassEnd,MassEnd};
    Double_t MeanMin[2]={MassStart,MassStart};
    Double_t GammaLim[2]={GammaStart,GammaEnd};
    Double_t GammaMin[2]={GammaStart,GammaStart};
    Double_t GammaMax[2]={GammaEnd,GammaEnd};

   //X-axises//
       TGraph* XGrid1 = new TGraph(2,MeanLim,y1);
       TGraph* XGrid2 = new TGraph(2,MeanLim,y2);
       TGraph* XGrid3 = new TGraph(2,MeanLim,y3);
       TGraph* XMin = new TGraph(2,MeanMin,GammaLim);
       TGraph* XMax = new TGraph(2,MeanMax,GammaLim);
	   
	    XGrid1->SetLineWidth(1);
       XGrid1->SetLineStyle(3);
       XGrid1->SetLineColor(31);

       XGrid3->SetLineWidth(1);
       XGrid3->SetLineStyle(3);
       XGrid3->SetLineColor(31);

       XGrid2->SetLineWidth(1);
       XGrid2->SetLineStyle(3);
       XGrid2->SetLineColor(31);

      
	   
	   XMin->SetLineWidth(1);
       XMin->SetLineStyle(3);
       XMin->SetLineColor(31);

       XMax->SetLineWidth(1);
       XMax->SetLineStyle(3);
       XMax->SetLineColor(31);

    //Y-axises//
       TGraph* YGrid1 = new TGraph(2,x1,GammaLim);
       TGraph* YMin = new TGraph(2,MeanLim,GammaMin);
       TGraph* YMax = new TGraph(2,MeanLim,GammaMax);
	    
		 YGrid1->SetLineWidth(1);
        YGrid1->SetLineStyle(3);
        YGrid1->SetLineColor(31);
	   
	    YMin->SetLineWidth(1);
       YMin->SetLineStyle(3);
       YMin->SetLineColor(31);
	   
	    YMax->SetLineWidth(1);
       YMax->SetLineStyle(3);
       YMax->SetLineColor(31);


////////////////////////////////////////////////////////////////////////////////////

       grid->Add(XGrid1,"L");
       grid->Add(XGrid2,"L");
       grid->Add(XGrid3,"L");
       grid->Add(YGrid1,"L");
       grid->Add(YMax,"L");
       grid->Add(YMin,"L");
       grid->Add(XMax,"L");
       grid->Add(XMin,"L");


       grid->Add(Start1,"P");
       grid->Add(Start2,"P");
       grid->Add(Start3,"P");
       grid->Add(Start4,"P");
       grid->Add(Start5,"P");
       grid->Add(Start6,"P");
       grid->Add(Start7,"P");
       grid->Add(Start8,"P");

#endif
////////////////////////////////////////////////////////////////////////////////////
   //TIME VARIABLES//
  TDatime *starttime = new TDatime();
  Int_t Date = starttime->GetDate();
  Int_t Clock = starttime->GetTime();
  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);

////////////////////////////////////////////////////////////////////////////////////
//RANDOMIZER FOR MULTIPLE PROCESS//
////////////////////////////////////////////////////////////////////////////////////

  timeval trand;
  gettimeofday(&trand,NULL);
  long int msRand = trand.tv_sec * 1000 + trand.tv_usec / 1000;
  TRandom3 fileran(msRand);
  Double_t randomize = fileran.Uniform(fileran.Uniform(fileran.Uniform(13.0)))+fileran.Uniform(2.0);
  rndInt *= (int)(100*randomize);

  //START TIME//
  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);

////////////////////////////////////////////////////////////////////////////////////
//FILES
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
//Fixed Parameters
  #ifdef GAMMAFIX
  gFix=1;
  #endif
  
  #ifdef MEANFIX
  mFix=1;
  #endif
  
  #ifdef NEGSIG
  sNeg=1;
  #endif

////////////////////////////////////////////////////////////////////////////////////  
  //CHIS TXT FILE
  //Organizing in Directories
  sprintf(dircname,"mkdir ./txt_files_imp/%d/",Date);
  system(dircname);
  sprintf(dircname,"mkdir ./txt_files_imp/%d/%d/",Date,(int)iter);
  system(dircname);
  sprintf(dircname,"mkdir ./txt_files_imp/%d/%d/%d%d%d/",Date,(int)iter,gFix,mFix,sNeg);
  system(dircname);
  
  #ifndef NLLBEST
  #ifdef READING
  sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCDeltaChisGoox%d%d%d-%d-%d-%d.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys);
  ofstream chiFile(filename);
  #else
  sprintf(filename,"txt_files_imp/%d-ToyMCDeltaChisGoox%d%d%d-%d-%d-%.2f.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,randomize);
  ofstream chiFile(filename);
  #endif
  #endif
  
  
  #ifdef NLLBEST
  #ifdef READING
  sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCDeltaChisGooNLLx%d%d%d-%d-%d-%d.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys);
  ofstream chiFileNLL(filename);
  //sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCNullNLLGoox%d%d%d-%d-%d-%d.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys);
  //ofstream nullNLLFile(filename);
  sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCNLLGoox%d%d%d-%d-%d-%d.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys);
  ofstream fileNLL(filename);
  #else
  sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCDeltaChisGooNLLx%d%d%d-%d-%d-%d-%.3f.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys,randomize);
  ofstream chiFileNLL(filename);
  //sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCNullNLLGoox%d%d%d-%d-%d-%d-%.3f.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys,randomize);
  //ofstream nullNLLFile(filename);
  sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCNLLGoox%d%d%d-%d-%d-%d-%.3f.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys,randomize);
  ofstream fileNLL(filename);
  #endif
  #endif
  //sprintf(filename,"txt_files_imp/%d-ChiCompare-%d.txt",iter,Clock);
  //ofstream chiCompare(filename);
  
////////////////////////////////////////////////////////////////////////////////////  
  //ROOT FILE
  
  sprintf(dircname,"mkdir ./Histos/%d/",Date);
  system(dircname);
  sprintf(dircname,"mkdir ./Histos/%d/%d/",Date,(int)iter);
  system(dircname);
  #ifdef READING
  sprintf(filename,"./Histos/%d/%d/%dRunsToyMCx%d%d%d-%d-%d-%d.root",Date,iter,iter,gFix,mFix,sNeg,Date,Clock,toys);
  #else
  sprintf(filename,"./Histos/%d/%d/%dRunsToyMCx%d%d%d-%d-%d-%d-%f.root",Date,iter,iter,gFix,mFix,sNeg,Date,Clock,toys,randomize);
  #endif
  sprintf(bufferstring,"ToyMC %d Runs ",iter);
  TFile GooFile(filename,"RECREATE",bufferstring);

////////////////////////////////////////////////////////////////////////////////////  
  //INPUT FILE
  #ifdef READING
  sprintf(filename,"./Input/20150425/%d/%dToysGenerated%d.root",iter,iter,toys);
  //sprintf(filename,"./Input/%d/%d/ToysGenerated%d.root",Date,iter,toys);
  TFile* fileInput = TFile::Open(filename);
  #endif
  
////////////////////////////////////////////////////////////////////////////////////
  //TOYS FILE
  #ifdef TOYWRITE
  sprintf(dircname,"mkdir ./Input/%d/",Date);
  system(dircname);
  
  sprintf(dircname,"mkdir ./Input/%d/%d/",Date,(int)iter);
  system(dircname);
		
  sprintf(filename,"./Input/%d/%d/%dRunsToysGenerated-%d-%f.root",Date,iter,iter,Date,randomize);
  sprintf(bufferstring,"ToyMC %d Runs Datasets",iter);
  TFile ToyGenFile(filename,"RECREATE",bufferstring);
  #endif

////////////////////////////////////////////////////////////////////////////////////
  //CANVAS
  TCanvas* canvas = new TCanvas("","",1200,1000);

////////////////////////////////////////////////////////////////////////////////////
  //DATASETS//
  BinnedDataSet dataNull(xvar);
  BinnedDataSet dataSig(xvar);

////////////////////////////////////////////////////////////////////////////////////
  //PHYSICS
  #ifdef GAMMAFIX
  Variable *Gamma = new Variable("Gamma",GAMMAFIX);
  #endif
  #ifndef GAMMAFIX
  Variable *Gamma = new Variable("Gamma",0.0150,0.00001,GammaStart,GammaEnd);
  #endif
  
  #ifndef MEANFIX
  Variable *Mean = new Variable("Mean",1.040,0.0001,MassStart,MassEnd);
  #endif
  #ifdef MEANFIX
  Variable *Mean = new Variable("Mean",MEANFIX);
  #endif
  
  Variable* Sigma = new Variable("Sigma", 0.002);

////////////////////////////////////////////////////////////////////////////////////
  //HISTOGRAMS//
  #ifndef READING
  TH1F* genHist = new TH1F("genHist", "",xvar->numbins, xvar->lowerlimit, xvar->upperlimit); //DATA HISTO
  #endif
  #ifdef READING
  TH1F* genHist;
  #endif
  
  TH1F pdfNullHist("pdfNullHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit); //NULL PDF HISTO
  TH1F pdfSigHist("pdfSignalHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  
  vector<TH1F> pdfSigHistos;
  TH1F pdfSig1Hist("pdfSig1Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig1Hist);
  TH1F pdfSig2Hist("pdfSig2Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig2Hist);
  TH1F pdfSig3Hist("pdfSig3Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig3Hist);
  TH1F pdfSig4Hist("pdfSig4Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig4Hist);
  TH1F pdfSig5Hist("pdfSig5Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig5Hist);
  TH1F pdfSig6Hist("pdfSig6Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig6Hist);
  TH1F pdfSig7Hist("pdfSig7Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig7Hist);
  TH1F pdfSig8Hist("pdfSig8Hist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfSigHistos.push_back(pdfSig8Hist);
  TH1F* pdfSigBestHist = new TH1F("pdfSigBestHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  
  #ifdef GOODPLOTS 
  //TH1F pdfNullHistPlot("pdfNullHistPlot", "",PLOTTINGFINENESS, xvar->lowerlimit, xvar->upperlimit); //NULL PDF HISTO
  //TH1F pdfSigHistPlot("pdfSigHistPlot", "",PLOTTINGFINENESS, xvar->lowerlimit, xvar->upperlimit);
  
	//BEST PARAMETERS TXT FILE
  sprintf(filename,"txt_files/%d-%d-ToyMCPlotParaMetersGoo-%d.txt",Date,Clock,iter);
  ofstream paramFile(filename);

  #endif
  
  #ifdef STARTINGPOINTS
  TH1I* startingPoints = new TH1I("Starting Points", "", 10,0,10);
  TH1I* startingPointsNLL = new TH1I("Starting Points Nll", "", 10,0,10);
  #endif
  
  #ifdef SCATTERPLOTS
  Int_t scatterBin = 100;
  Double_t gammaLow = GammaStart - GammaStart*0.5;
  Double_t gammaHigh = GammaEnd + GammaStart*0.5;
  Double_t meanLow = MassStart-MassStart*0.001;
  Double_t meanHigh = MassEnd+MassStart*0.001;
  
//////////////////////////////////////////////////////////////////////////////////////////////////////////
//TOTAL SCATTER PLOTS////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////
//CHI vs SIG
  
  TH2F scatterPlotChiSigBest("scatterSigBest","Chi vs Signal Fraction",100,0,0.05,100,0,30);
  TH2F scatterPlotChiSigBestSLim("scatterPlotChiSigBestSLim","Chi vs Signal Fraction",100,0,0.05,100,0,30);
//CHI vs MEAN
 
  TH2F scatterPlotChiMeanBest("scatterPlotChiMeanBest","Chi vs Mean ",100,1.035,1.065,100,0,30);
  TH2F scatterPlotChiMeanBestSLim("scatterPlotChiMeanBestSLim","Chi vs Mean ",100,1.035,1.065,100,0,30);
  TH2F scatterPlotChiMeanBestMLim("scatterPlotChiMeanBestMLim","Chi vs Mean ",100,1.035,1.065,100,0,30);
  TH2F scatterPlotChiMeanBestGLim("scatterPlotChiMeanBestGLim","Chi vs Mean ",100,1.035,1.065,100,0,30);
//CHI vs GAMMA
  
  TH2F scatterPlotChiGammaBest("scatterPlotChiGammaBest","Chi vs Gamma ",100,0.001,0.07,100,0,30);
  TH2F scatterPlotChiGammaBestSLim("scatterPlotChiGammaBestSLim","Chi vs Gamma ",100,0.001,0.07,100,0,30);
  TH2F scatterPlotChiGammaBestMLim("scatterPlotChiGammaBestMLim","Chi vs Gamma ",100,0.001,0.07,100,0,30);
  TH2F scatterPlotChiGammaBestGLim("scatterPlotChiGammaBestGLim","Chi vs Gamma ",100,0.001,0.07,100,0,30);
//SIG vs MEAN

  TH2F scatterPlotSigMeanBest("scatterPlotSigMeanBest","Signal Fraction vs Mean ",100,1.035,1.065,100,0,0.05);
  TH2F scatterPlotSigMeanBestSLim("scatterPlotSigMeanBestSLim","Signal Fraction vs Mean ",100,1.035,1.065,100,0,0.05);
  TH2F scatterPlotSigMeanBestMLim("scatterPlotSigMeanBestMLim","Signal Fraction vs Mean ",100,1.035,1.065,100,0,0.05);
//SIG vs GAMMA

  TH2F scatterPlotSigGammaBest("scatterPlotSigGammaBest","Signal Fraction vs Gamma",100,0.001,0.07,100,0,0.05);
  TH2F scatterPlotSigGammaBestSLim("scatterPlotSigGammaBestSLim","Signal Fraction vs Gamma",100,0.001,0.07,100,0,0.05);
  TH2F scatterPlotSigGammaBestGLim("scatterPlotSigGammaBestGLim","Signal Fraction vs Gamma",100,0.001,0.07,100,0,0.05);
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//STARTING POINTS SCATTER PLOTS GAMMA -MEAN ///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  
  vector<TH1F> deltaChiStarts;
  TH1F ToyDelta1("ToyDelta1","Toy MC Delta Chi Square 1",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta1);
  TH1F ToyDelta2("ToyDelta2","Toy MC Delta Chi Square 2",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta2);
  TH1F ToyDelta3("ToyDelta3","Toy MC Delta Chi Square 3",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta3);
  TH1F ToyDelta4("ToyDelta4","Toy MC Delta Chi Square 4",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta4);
  TH1F ToyDelta5("ToyDelta5","Toy MC Delta Chi Square 5",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta5);
  TH1F ToyDelta6("ToyDelta6","Toy MC Delta Chi Square 6",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta6);
  TH1F ToyDelta7("ToyDelta7","Toy MC Delta Chi Square 7",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta7);
  TH1F ToyDelta8("ToyDelta8","Toy MC Delta Chi Square 8",5,0,0.0001);
  deltaChiStarts.push_back(ToyDelta8);
  
  vector<TH2F> scatterPlotGammaMean;
  TH2F scatterPlotGammaMean1("scatterPlotGammaMean1","Gamma vs Mean Start 1",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean1);
  TH2F scatterPlotGammaMean2("scatterPlotGammaMean2","Gamma vs Mean Start 2",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean2);
  TH2F scatterPlotGammaMean3("scatterPlotGammaMean3","Gamma vs Mean Start 3",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean3);
  TH2F scatterPlotGammaMean4("scatterPlotGammaMean4","Gamma vs Mean Start 4",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean4);
  TH2F scatterPlotGammaMean5("scatterPlotGammaMean5","Gamma vs Mean Start 5",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean5);
  TH2F scatterPlotGammaMean6("scatterPlotGammaMean6","Gamma vs Mean Start 6",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean6);
  TH2F scatterPlotGammaMean7("scatterPlotGammaMean7","Gamma vs Mean Start 7",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean7);
  TH2F scatterPlotGammaMean8("scatterPlotGammaMean8","Gamma vs Mean Start 8",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMean.push_back(scatterPlotGammaMean8);
  
  vector<TH2F> scatterPlotGammaMeanSLim;
  TH2F scatterPlotGammaMean1SLim("scatterPlotGammaMean1SLim","Gamma vs Mean 1 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean1SLim);
  TH2F scatterPlotGammaMean2SLim("scatterPlotGammaMean2SLim","Gamma vs Mean 2 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean2SLim);
  TH2F scatterPlotGammaMean3SLim("scatterPlotGammaMean3SLim","Gamma vs Mean 3 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean3SLim);
  TH2F scatterPlotGammaMean4SLim("scatterPlotGammaMean4SLim","Gamma vs Mean 4 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean4SLim);
  TH2F scatterPlotGammaMean5SLim("scatterPlotGammaMean5SLim","Gamma vs Mean 5 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean5SLim);
  TH2F scatterPlotGammaMean6SLim("scatterPlotGammaMean6SLim","Gamma vs Mean 6 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean6SLim);
  TH2F scatterPlotGammaMean7SLim("scatterPlotGammaMean7SLim","Gamma vs Mean 7 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean7SLim);
  TH2F scatterPlotGammaMean8SLim("scatterPlotGammaMean8SLim","Gamma vs Mean 8 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanSLim.push_back(scatterPlotGammaMean8SLim);
   
   
  vector<TH2F> scatterPlotGammaMeanMLim;
  TH2F scatterPlotGammaMean1MLim("scatterPlotGammaMean1MLim","Gamma vs Mean 1 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean1MLim);
  TH2F scatterPlotGammaMean2MLim("scatterPlotGammaMean2MLim","Gamma vs Mean 2 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean2MLim);
  TH2F scatterPlotGammaMean3MLim("scatterPlotGammaMean3MLim","Gamma vs Mean 3 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean3MLim);
  TH2F scatterPlotGammaMean4MLim("scatterPlotGammaMean4MLim","Gamma vs Mean 4 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean4MLim);
  TH2F scatterPlotGammaMean5MLim("scatterPlotGammaMean5MLim","Gamma vs Mean 5 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean5MLim);
  TH2F scatterPlotGammaMean6MLim("scatterPlotGammaMean6MLim","Gamma vs Mean 6 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean6MLim);
  TH2F scatterPlotGammaMean7MLim("scatterPlotGammaMean7MLim","Gamma vs Mean 7 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean7MLim);
  TH2F scatterPlotGammaMean8MLim("scatterPlotGammaMean8MLim","Gamma vs Mean 8 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanMLim.push_back(scatterPlotGammaMean8MLim);
  
  vector<TH2F> scatterPlotGammaMeanGLim;
  TH2F scatterPlotGammaMean1GLim("scatterPlotGammaMean1GLim","Gamma vs Mean 1 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean1GLim);
  TH2F scatterPlotGammaMean2GLim("scatterPlotGammaMean2GLim","Gamma vs Mean 2 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean2GLim);
  TH2F scatterPlotGammaMean3GLim("scatterPlotGammaMean3GLim","Gamma vs Mean 3 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean3GLim);
  TH2F scatterPlotGammaMean4GLim("scatterPlotGammaMean4GLim","Gamma vs Mean 4 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean4GLim);
  TH2F scatterPlotGammaMean5GLim("scatterPlotGammaMean5GLim","Gamma vs Mean 5 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean5GLim);
  TH2F scatterPlotGammaMean6GLim("scatterPlotGammaMean6GLim","Gamma vs Mean 6 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean6GLim);
  TH2F scatterPlotGammaMean7GLim("scatterPlotGammaMean7GLim","Gamma vs Mean 7 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean7GLim);
  TH2F scatterPlotGammaMean8GLim("scatterPlotGammaMean8GLim","Gamma vs Mean 8 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanGLim.push_back(scatterPlotGammaMean8GLim);
  
  vector<TH2F> scatterPlotGammaMeanBest;
  TH2F scatterPlotGammaMean1Best("scatterPlotGammaMean1Best","Gamma vs Mean 1 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean1Best);
  TH2F scatterPlotGammaMean2Best("scatterPlotGammaMean2Best","Gamma vs Mean 2 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean2Best);
  TH2F scatterPlotGammaMean3Best("scatterPlotGammaMean3Best","Gamma vs Mean 3 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean3Best);
  TH2F scatterPlotGammaMean4Best("scatterPlotGammaMean4Best","Gamma vs Mean 4 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean4Best);
  TH2F scatterPlotGammaMean5Best("scatterPlotGammaMean5Best","Gamma vs Mean 5 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean5Best);
  TH2F scatterPlotGammaMean6Best("scatterPlotGammaMean6Best","Gamma vs Mean 6 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean6Best);
  TH2F scatterPlotGammaMean7Best("scatterPlotGammaMean7Best","Gamma vs Mean 7 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean7Best);
  TH2F scatterPlotGammaMean8Best("scatterPlotGammaMean8Best","Gamma vs Mean 8 Best",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBest.push_back(scatterPlotGammaMean8Best);
  
  
  vector<TH2F> scatterPlotGammaMeanBestSLim;
  TH2F scatterPlotGammaMean1BestSLim("scatterPlotGammaMean1BestSLim","Gamma vs Mean Best 1 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean1BestSLim);
  TH2F scatterPlotGammaMean2BestSLim("scatterPlotGammaMean2BestSLim","Gamma vs Mean Best 2 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean2BestSLim);
  TH2F scatterPlotGammaMean3BestSLim("scatterPlotGammaMean3BestSLim","Gamma vs Mean Best 3 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean3BestSLim);
  TH2F scatterPlotGammaMean4BestSLim("scatterPlotGammaMean4BestSLim","Gamma vs Mean Best 4 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean4BestSLim);
  TH2F scatterPlotGammaMean5BestSLim("scatterPlotGammaMean5BestSLim","Gamma vs Mean Best 5 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean5BestSLim);
  TH2F scatterPlotGammaMean6BestSLim("scatterPlotGammaMean6BestSLim","Gamma vs Mean Best 6 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean6BestSLim);
  TH2F scatterPlotGammaMean7BestSLim("scatterPlotGammaMean7BestSLim","Gamma vs Mean Best 7 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean7BestSLim);
  TH2F scatterPlotGammaMean8BestSLim("scatterPlotGammaMean8BestSLim","Gamma vs Mean Best 8 SLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestSLim.push_back(scatterPlotGammaMean8BestSLim);
  
  vector<TH2F> scatterPlotGammaMeanBestMLim;
  TH2F scatterPlotGammaMean1BestMLim("scatterPlotGammaMean1BestMLim","Gamma vs Mean Best 1 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean1BestMLim);
  TH2F scatterPlotGammaMean2BestMLim("scatterPlotGammaMean2BestMLim","Gamma vs Mean Best 2 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean2BestMLim);
  TH2F scatterPlotGammaMean3BestMLim("scatterPlotGammaMean3BestMLim","Gamma vs Mean Best 3 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean3BestMLim);
  TH2F scatterPlotGammaMean4BestMLim("scatterPlotGammaMean4BestMLim","Gamma vs Mean Best 4 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean4BestMLim);
  TH2F scatterPlotGammaMean5BestMLim("scatterPlotGammaMean5BestMLim","Gamma vs Mean Best 5 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean5BestMLim);
  TH2F scatterPlotGammaMean6BestMLim("scatterPlotGammaMean6BestMLim","Gamma vs Mean Best 6 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean6BestMLim);
  TH2F scatterPlotGammaMean7BestMLim("scatterPlotGammaMean7BestMLim","Gamma vs Mean Best 7 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean7BestMLim);
  TH2F scatterPlotGammaMean8BestMLim("scatterPlotGammaMean8BestMLim","Gamma vs Mean Best 8 MLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestMLim.push_back(scatterPlotGammaMean8BestMLim);
  
  vector<TH2F> scatterPlotGammaMeanBestGLim;
  TH2F scatterPlotGammaMean1BestGLim("scatterPlotGammaMean1BestGLim","Gamma vs Mean Best 1 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean1BestGLim);
  TH2F scatterPlotGammaMean2BestGLim("scatterPlotGammaMean2BestGLim","Gamma vs Mean Best 2 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean2BestGLim);
  TH2F scatterPlotGammaMean3BestGLim("scatterPlotGammaMean3BestGLim","Gamma vs Mean Best 3 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean3BestGLim);
  TH2F scatterPlotGammaMean4BestGLim("scatterPlotGammaMean4BestGLim","Gamma vs Mean Best 4 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean4BestGLim);
  TH2F scatterPlotGammaMean5BestGLim("scatterPlotGammaMean5BestGLim","Gamma vs Mean Best 5 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean5BestGLim);
  TH2F scatterPlotGammaMean6BestGLim("scatterPlotGammaMean6BestGLim","Gamma vs Mean Best 6 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean6BestGLim);
  TH2F scatterPlotGammaMean7BestGLim("scatterPlotGammaMean7BestGLim","Gamma vs Mean Best 7 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean7BestGLim);
  TH2F scatterPlotGammaMean8BestGLim("scatterPlotGammaMean8BestGLim","Gamma vs Mean Best 8 GLim",scatterBin,meanLow,meanHigh,scatterBin,gammaLow,gammaHigh);
  scatterPlotGammaMeanBestGLim.push_back(scatterPlotGammaMean8BestGLim);
  #endif
 
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//DELTA CHI SQUARE HISTO////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
  TH1F ToyMCDeltaChisquare("myToyLocalDeltaChiSqua  re","Toy MC Delta Chi Square Distribution",600,0,60);
  ToyMCDeltaChisquare.GetXaxis()->SetTitle("#Delta#chi^{2}  (Toy MC)");

////////////////////////////////////////////////////////////////////////////////////
  //SIGNAL FRACTIONS//
  Variable *nB = new Variable("nBkg",events,0,1E4); 
  #ifdef NEGSIG
  Variable *sFrac = new Variable("fSig",0.,0.001,-NEGSIG,NEGSIG);
  #endif
  #ifndef NEGSIG
  Variable *sFrac = new Variable("fSig",0.,0.001,0.0,0.25);
  #endif
  //Variable *sfracB = new Variable("f_best",0,0.001,0.0,1.0);


////////////////////////////////////////////////////////////////////////////////////
  //CHI SQUARES & NLL
  Double_t NullFitChiSq;

  //Nll
  Double_t NullNLL;


////////////////////////////////////////////////////////////////////////////////////
  //EVALUATION VECTORS
  vector<Double_t> ValsNull;
  vector<Double_t> ValsTot;
  vector<Double_t> ValsSig;
  vector<Variable*> vars;
  vector<PdfBase*> comps;

////////////////////////////////////////////////////////////////////////////////////
//COUNTERS
  int cycles=0; //TOTAL CYCLES
  int nfits=0;  //total number of fits
  int sectionCounter = 0;
  Double_t totalpdf=0;
  
  int problemFit[72]={9,13,31,58,87,101,102,112,119,124,139,142,150,152,173,182,205,218,224,232,235,274,316,319,324,328,332,340,355,369,374,379,405,422,440,444,479,481,505,508,509,511,516,528,555,586,627,639,646,651,653,663,679,684,685,692,706,710,763,792,797,801,812,825,837,845,875,892,911,914,926,947};
  
for(int l=0;l<iter;l++){

  cout<<endl;
  cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
  cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Iteration "<<l<<" ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
  cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
  cout<<endl;

////////////////////////////////////////////////////////////////////////////////////
//INITIALIZE COUNTERS
  ++cycles;

#ifndef READING
////////////////////////////////////////////////////////////////////////////////////
//INITIALIZE DATA HIST
  for (int i = 0; i <= xvar->numbins+1; ++i) {
   genHist->SetBinContent(i,0);
 }
#endif
////////////////////////////////////////////////////////////////////////////////////
//SETTING STARTING VALUES for Bkg

  nB->error=0.5;
  nB->value=2500;

////////////////////////////////////////////////////////////////////////////////////
//SEED//
  struct timeval tp;
  gettimeofday(&tp,NULL);
  long int ms = tp.tv_sec * 1000 + tp.tv_usec / 1000;
  //cout<<"Milliseconds"<<ms<<endl;
////////////////////////////////////////////////////////////////////////////////////
//RANDOM GENERATOR//
  TRandom donram(ms+rndInt);

#ifdef READING   
////////////////////////////////////////////////////////////////////////////////////
//READING EVENTS 
////////////////////////////////////////////////////////////////////////////////////
//FILLING DATA HISTO
	
	//DATA TXT FILE
	//sprintf(filename,"./txt_files_imp/%d-ToyGenerated-10000/%d-ToyMCGenrated_%d.txt",section,section,sectionCounter);
	
	cout<<"==============================================================================="<<endl;
	cout<<"READING FILE : "<<filename<<endl;
	cout<<"==============================================================================="<<endl;
    
	sprintf(histoname,"genHist%d",l);
	sprintf(bufferstring,"genHist%d",l);

	genHist = (TH1F*)fileInput->Get(histoname);

	GooFile.cd();
	
	#ifdef TOYWRITE
	ToyGenFile.cd();
    genHist->Write();
    GooFile.cd();
    #endif
	
	
#else
////////////////////////////////////////////////////////////////////////////////////
//GENERATING EVENTS - HIT & MISS
  Double_t roll=0;
  Double_t background=0;
  
	#ifdef TOYWRITE
	
	//sprintf(filename,"txt_files_imp/%d-%dToyMCDeltaChisGooNLL-%f-%d.txt",iter,Clock,randomize,l);
	//ofstream toyGenerated(filename);
	
	#endif

   //#pragma omp parallel for
  for (int j = 0; j < events; ++j) {
    xvar->value = donram.Uniform(0.56)+1.008;
    background = fondo(xvar->value);
    roll = donram.Uniform(10);
    if (roll > background) {
      --j;
      continue; }

    if(isnan(background)){ //CHECKING NAN PROBLEMS
     --j;
     continue;}

    if ((xvar->value < xvar->lowerlimit) || (xvar->value > xvar->upperlimit)) {
       --j;
      continue;}

      genHist->Fill(xvar->value);
	  #ifdef TOYWRITE
	  //toyGenerated<<xvar->value;
	  #endif
  }
  
  #ifdef TOYWRITE
  
  ToyGenFile.cd();
  sprintf(bufferstring,"genHist%d",l);	
  genHist->SetTitle(bufferstring);
  genHist->SetName(bufferstring);
  genHist->Write();
  GooFile.cd();
  #endif
  
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////
//FAKE TEST
//genHist->SetBinContent(2,genHist->GetBinContent(2)+500);
//genHist->SetBinContent(20,genHist->GetBinContent(20)-500);
//////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////////////
//FILLING THE DATASETS
//////////////////////////////////////////////////////////////////////////////////////////////////

  for (int i = 1; i <= xvar->numbins; ++i) {
   dataNull.setBinContent(i-1,genHist->GetBinContent(i));
   dataSig.setBinContent(i-1,genHist->GetBinContent(i));
 }

//////////////////////////////////////////////////////////////////////////////////////////////////
//BKG PDF
//////////////////////////////////////////////////////////////////////////////////////////////////

  ThreePdf ThreePdfB("Three Bkg",xvar);
  GooPdf* ThreePdfBPtr = &ThreePdfB; 

//////////////////////////////////////////////////////////////////////////////////////////////////
//NULL PDF
//////////////////////////////////////////////////////////////////////////////////////////////////


 comps.push_back(ThreePdfBPtr);
 vars.push_back(nB);
 AddPdf NullPdf ("Three Bodies Bkg",vars,comps,1);
 GooPdf* NullPdfPtr = &NullPdf;
 vars.clear();
 comps.clear();

//////////////////////////////////////////////////////////////////////////////////////////////////
//NULL FIT
//////////////////////////////////////////////////////////////////////////////////////////////////

 NullPdfPtr->setData(&dataNull);
 NullPdfPtr->setFitControl(new BinnedNllFitInt());

 FitManager fitterNull(NullPdfPtr);
 fitterNull.fit(); nfits++; //NFITS JUST COUNTS THE NUMBER OF FITS

 fitterNull.getMinuitValues();

//////////////////////////////////////////////////////////////////////////////////////////////////
//TMINUIT NULL & MINOS
//////////////////////////////////////////////////////////////////////////////////////////////////

  TMinuit* minuNull= fitterNull.getMinuitObject();
  //minuNull->mnhess(); nfits++;  //PERFROMING HESSE
  //minuNull->mnmnos(); nfits++; //PERFROMING MINOS

  //GETTING NULL VALUES
  NullPdfPtr->evaluateAtPointsInt(xvar,ValsNull);

Double_t totalpdf = 0;

//#pragma omp parallel for
for(int k=0;k<xvar->numbins;k++){
        	
        pdfNullHist.SetBinContent(k+1,ValsNull[k]);
        totalpdf += ValsNull[k];
  	//cout<<"PDF BKG= "<<ValsNull[k]<<endl;
}
//cout<<"Total PDF= "<<totalpdf<<endl;
Double_t eventipdf=0;

//#pragma omp parallel for
for(int k=0;k<xvar->numbins;k++){

Double_t val = pdfNullHist.GetBinContent(k+1);
    val /= totalpdf;
    val *=events;
    pdfNullHist.SetBinContent(k+1, val);
    eventipdf+=val;
}

ValsNull.clear();

NullFitChiSq = chisquare(*genHist,pdfNullHist,xvar);

#ifdef DEBUGGINGCOUTS
cout<<"=================================================================================="<<endl;
cout << "CHI SQUARE BKG PDF "<<NullFitChiSq<<endl;
cout<<"=================================================================================="<<endl;
#endif

 Double_t* NullValues;
 NullValues=(minuNull->fU); //VARIABLE VALUES
 Double_t NullValues1[1];
 NullValues1[0]=NullValues[0];
 Double_t NullBkgLim=TMath::Abs(TMath::Cos(minuNull->fX[minuNull->fNiofex[0]-1])); //FSIG LIMIT
 
 NullNLL = minuNull->fAmin;

//////////////////////////////////////////////////////////////////////////////////////////////////
///// SIGNAL FIT /////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

  //Double_t LarghStart[4]={0.005,0.0235,0.0420,0.0605};
  #ifndef GAMMAFIX
  Double_t LarghStart[4]={0.015,0.0235,0.0420,0.0605};
  #endif
  Double_t MassaStart[2]={1.040,1.050};

  Double_t SigValues[8][3];
  Double_t SigErrors[8][3];
  Double_t SignalChiSquare[8];
  Double_t SignalNLL[8];
  
  Double_t SignalLimit[8];
  Double_t MeanLimit[8];
  Double_t GammaLimit[8];

  int fitCounter = 0;

#ifndef GAMMAFIX
 for(int gscans = 0;gscans<4;gscans++){
	 
        cout<<endl;
        cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
        cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Gamma "<<gscans+1<<"("<<l<<")~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
#endif
#ifndef MEANFIX
		for(int mscans = 0;mscans<2;mscans++){
        cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
        cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Mass "<<mscans+1<<" ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
        cout<<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"<<endl;
        cout<<endl;
        cout<<endl;
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////
//SETTING STARTING VALUES
//////////////////////////////////////////////////////////////////////////////////////////////////

  sFrac->value=0.001;
  sFrac->error=5E-4;
  
  #ifndef GAMMAFIX
  Gamma->value=LarghStart[gscans];
  Gamma->error=1E-4;
  #endif
  #ifdef GAMMAFIX
  Gamma->value=GAMMAFIX;
  int gscans = 0;
  #endif
  
  #ifndef MEANFIX
  Mean->value=MassaStart[mscans];	
  Mean->error=2E-3;
  #endif
  
  #ifdef MEANFIX
  Mean->value=MEANFIX;	
  int mscans = 0;
  #endif
 
//////////////////////////////////////////////////////////////////////////////////////////////////
//BKG PDF
//////////////////////////////////////////////////////////////////////////////////////////////////
 
 sprintf(pdfname,"Bkg Comp - %d - %d -%d",l,gscans,mscans);
 ThreePdf ThreePdfS(pdfname,xvar);
 GooPdf* ThreePdfSPtr = &ThreePdfS; 

//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////
//SIGNAL PDF

 //GooPdf* AllPdfsSig1 = new VoigtianPdf("Bw PDF1", xvar, Mean,Sigma,Gamma);
 sprintf(pdfname,"Peak Comp - %d - %d -%d",l,gscans,mscans);
 VoigtianThreshPdf PeakPdf(pdfname, xvar, Mean,Sigma,Gamma);
 GooPdf* PeakPdfPtr = &PeakPdf;

//////////////////////////////////////////////////////////////////////////////////////////////////
 sprintf(pdfname,"Total Pdf - %d - %d -%d",l,gscans,mscans);
 AddPdf TotalPdfSig(pdfname,sFrac,PeakPdfPtr,ThreePdfSPtr,1);
 GooPdf* TotalPdfSigPtr = &TotalPdfSig; 

//////////////////////////////////////////////////////////////////////////////////////////////////
////SIGNAL FIT (MIGRAD)

 TotalPdfSigPtr->setData(&dataSig);
 TotalPdfSigPtr->setFitControl(new BinnedNllFitInt());
 FitManager fitterSig(TotalPdfSigPtr);

////////////////////////////////////////////////////////////////////////////////////
  //SIGNAL TMINUIT OBJECTS
  TMinuit* minuSig;

//////////////////////////////////////////////////////////////////////////////////////////////////
//SETTING MIGRAD FIT

  fitterSig.fit(); nfits++;
  fitterSig.getMinuitValues();
  minuSig = fitterSig.getMinuitObject();

//////////////////////////////////////////////////////////////////////////////////////////////////
//HESSE FIT
 //minuSig1->mnhess(); nfits++;

//////////////////////////////////////////////////////////////////////////////////////////////////
//TMINUIT SIGNAL
//////////////////////////////////////////////////////////////////////////////////////////////////

 Double_t* SigVal;
 Double_t* SigErr;

 SigVal=(minuSig->fU); //VARIABLE VALUES
 SigErr=(minuSig->fWerr); //VARIABLE ERRORS
 
 SigValues[fitCounter][0]=SigVal[0]; //fSig
 SigValues[fitCounter][1]=SigVal[1]; //Mean
 SigValues[fitCounter][2]=SigVal[3]; //Gamma
 
 SigErrors[fitCounter][0]= SigErr[0];
 SigErrors[fitCounter][1]= SigErr[1]; 
 SigErrors[fitCounter][2]= SigErr[3];
 
 SignalLimit[fitCounter]=TMath::Abs(TMath::Cos(minuSig->fX[minuSig->fNiofex[0]-1])); // fSig limit
 MeanLimit[fitCounter]=TMath::Abs(TMath::Cos(minuSig->fX[minuSig->fNiofex[1]-1]));   // Mean limit
 GammaLimit[fitCounter]=TMath::Abs(TMath::Cos(minuSig->fX[minuSig->fNiofex[3]-1]));  // Gamma limit
 

//////////////////////////////////////////////////////////////////////////////////////////////////
//SIGNAL PARAMETERS VALUES
//////////////////////////////////////////////////////////////////////////////////////////////////

 TotalPdfSigPtr->evaluateAtPointsInt(xvar,ValsSig);

 totalpdf = 0;

//#pragma omp parallel for
for(int k=0;k<xvar->numbins;k++){
        pdfSigHistos[fitCounter].SetBinContent(k+1,ValsSig[k]);
        totalpdf += ValsSig[k];
}

//#pragma omp parallel for
for(int k=0;k<xvar->numbins;k++){

Double_t val = pdfSigHistos[fitCounter].GetBinContent(k+1);
    val /= totalpdf;
    val *= events;
    pdfSigHistos[fitCounter].SetBinContent(k+1, val);
}
    totalpdf = 0;
    ValsSig.clear();
//////////////////////////////////////////////////////////////////////////////////////////////////
//SIG CHI SQUARE -SIG NLL
//////////////////////////////////////////////////////////////////////////////////////////////////

//#ifndef NLLBEST
SignalChiSquare[fitCounter] = chisquare(*genHist,pdfSigHistos[fitCounter],xvar);
if(abs(NullFitChiSq-SignalChiSquare[fitCounter])<=5E-5){
	double BestChi=SignalChiSquare[fitCounter];
	for(int k=0;k<8;k++) SignalChiSquare[k]=BestChi;
	cout<<"=================================================================================="<<endl;
	cout<<"==============  JUMPING =========================================================="<<endl;
	cout<<"=================================================================================="<<endl;
	break;
	
} 
//#endif
//chisquare1= genHist->Chi2Test(&pdfSig1Hist,"CHI2UW");

SignalNLL[fitCounter] = minuSig->fAmin;

#ifdef DEBUGGINGCOUTS
#ifndef NLLBEST
cout<<"=================================================================================="<<endl;
cout<<"CHI SIG CHI SQUARE = "<<SignalChiSquare[fitCounter]<<endl;
cout<<"=================================================================================="<<endl;
#endif
#endif

//////////////////////////////////////////////////////////////////////////////////////////////////
//CLEANING SIGNAL FITS

    TotalPdfSigPtr->clearCurrentFit();
    PeakPdfPtr->clearCurrentFit();
    ThreePdfSPtr->clearCurrentFit();
    minuSig->mncler();
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	++fitCounter;

 #ifndef MEANFIX
 }

 if(abs(NullFitChiSq-SignalChiSquare[fitCounter])<=5E-5){
        double BestChi=SignalChiSquare[fitCounter];
        for(int k=0;k<8;k++) SignalChiSquare[k]=BestChi;
        cout<<"=================================================================================="<<endl;
        cout<<"==============  JUMPING =========================================================="<<endl;
        cout<<"=================================================================================="<<endl;
        break;

}

 #endif
 #ifndef GAMMAFIX
 }
 #endif

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//BEST CHI SQUARE-NLL BUFFERS & COUNTERS
Double_t bestChiSquare = -2.0;
Double_t bestChiSquareNLL = -2.0;
Double_t bestChi = -2.0;
Double_t bestNLL = 1000.0;
Double_t NLLChiSquare = 0;

int BestFitNum = 0;
int BestFitNumNLL = 0;
int BestFit = 0;
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//BEST CHI SELECTION
#ifndef NLLBEST
for (int p=0; p<fitCounter; p++) {
	
	if(bestChiSquare < NullFitChiSq-SignalChiSquare[p] && NullFitChiSq-SignalChiSquare[p]>=0){ bestChiSquare = approximate(NullFitChiSq-SignalChiSquare[p]); BestFitNum = p;}
				
	cout<<"----- Signal Chi Int "<<p+1<<" = "<<SignalChiSquare[p]<<"----- Best Chi Int = "<<bestChiSquare<<endl;
            }
	ToyMCDeltaChisquare.Fill(bestChiSquare);
	bestChi = bestChiSquare;
	BestFit = BestFitNum;
	chiFile<<bestChiSquare<<endl;//"  "<<BestFitNum<<endl;
	
	/*
	if(bestChiSquare<=1E-5){
		
		for(int j=0;j<8;j++){
			deltaChiStarts[j].Fill(approximate(NullFitChiSq-SignalChiSquare[j]));
		}
		
	}
	 */
	 
	//cout<<"Likelihood HISTOs"<<endl;
	double likeliHoodNull = 0.0;
	double likeliHoodSignal = 0.0;
	
	for(int k = 1;k<29;k++){
		double termineSig = (pdfSigHistos[BestFitNum].GetBinContent(k));
		double termineNull = pdfNullHist.GetBinContent(k);
		
		likeliHoodSignal += -2*genHist->GetBinContent(k)*log(termineSig);
		likeliHoodNull += -2*genHist->GetBinContent(k)*log(termineNull);
	}
	
	fileNLL<<likeliHoodNull-likeliHoodSignal<<endl;
	
#endif

#ifdef NLLBEST
for (int p=0; p<fitCounter; p++) {
	
	if(bestNLL > SignalNLL[p]){ bestNLL = SignalNLL[p]; BestFitNumNLL = p;}
            }
			
	NLLChiSquare = chisquare(*genHist,pdfSigHistos[BestFitNumNLL],xvar);
	bestChiSquareNLL = approximate(NullFitChiSq-NLLChiSquare);
	Double_t deltaChiNLL = bestChiSquareNLL-bestChiSquare;

	ToyMCDeltaChisquare.Fill(bestChiSquareNLL);
	
	bestChi = bestChiSquareNLL;
	BestFit = BestFitNumNLL;
	chiFileNLL<<bestChiSquareNLL<<endl; //"  "<<deltaChiNLL<<"  "<<BestFitNumNLL<<endl;
	//nullNLLFile<<NullNLL<<endl;
	//fileNLL<<bestNLL<<endl;
/*	
	if(bestChiSquareNLL<=1E-5){
		
		for(int j=0;j<8;j++){
			deltaChiStarts[j].Fill(approximate(NullFitChiSq-chisquare(*genHist,pdfSigHistos[j],xvar)));
		}
		
	}
*/	

	double likeliHoodNull = 0.0;
	double likeliHoodSignal = 0.0;
	
	for(int k = 1;k<29;k++){
		double termineSig = (pdfSigHistos[BestFitNum].GetBinContent(k));
		double termineNull = pdfNullHist.GetBinContent(k);
		
		likeliHoodSignal += -2*genHist->GetBinContent(k)*log(termineSig);
		likeliHoodNull += -2*genHist->GetBinContent(k)*log(termineNull);
	}
	
	fileNLL<<likeliHoodNull-likeliHoodSignal<<endl;
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//CHI COMPARE
/*
	Double_t maxDifference = 0.0;
	chiCompare<<"Best = "<<BestFitNum<<"List -->";
	
	for (int p=0; p<fitCounter; p++) {
		chiCompare<<p<<"  CHI =  "<<SignalChiSquare[p]<<"  NLL = "<<SignalNLL[p];
	}

	for (int p=0; p<fitCounter; p++) {
		for (int k=0; k<fitCounter; k++) 
			if(maxDifference<abs(SignalChiSquare[p]-SignalChiSquare[k])) maxDifference = SignalChiSquare[p]-SignalChiSquare[k];
			}
			 * 
	chiCompare<<" ---- Best Chi NLL  "<<BestFitNumNLL<<"  "<<NLLChiSquare<<" max diff = "<<maxDifference<<endl;
*/
	
    #ifdef STARTINGPOINTS
	#ifdef NLLBEST
	if(bestChi<=1E-5){
	startingPointsNLL->Fill(BestFitNumNLL);
	}
	#endif
	#ifndef NLLBEST
	if(bestChi<=1E-5){
	startingPoints->Fill(BestFitNum);
	}
	#endif
	#endif
	

#ifdef SCATTERPLOTS
#ifndef NLLBEST
Int_t flag = BestFitNum;
Double_t bestChiScatter = bestChiSquare;
#endif
#ifdef NLLBEST
Int_t flag = BestFitNumNLL;
Double_t bestChiScatter = bestChiSquareNLL;
#endif

	if(MeanLimit[flag]>=0.001 && GammaLimit[flag]>=0.001 && SignalLimit[flag]>=0.001){
    scatterPlotChiSigBest.Fill(SigValues[flag][0],bestChiScatter);
    scatterPlotChiGammaBest.Fill(SigValues[flag][2],bestChiScatter);
    scatterPlotChiMeanBest.Fill(SigValues[flag][1],bestChiScatter);
    scatterPlotSigMeanBest.Fill(SigValues[flag][1],SigValues[flag][0]);
    scatterPlotSigGammaBest.Fill(SigValues[flag][2],SigValues[flag][0]);
    scatterPlotGammaMeanBest[flag].Fill(SigValues[flag][1],SigValues[flag][2]);
        }

    if(SignalLimit[flag]<0.001){
    scatterPlotChiSigBestSLim.Fill(SigValues[flag][0],bestChiScatter);
    scatterPlotChiGammaBestSLim.Fill(SigValues[flag][2],bestChiScatter);
    scatterPlotChiMeanBestSLim.Fill(SigValues[flag][1],bestChiScatter);
    scatterPlotSigMeanBestSLim.Fill(SigValues[flag][1],SigValues[flag][0]);
    scatterPlotSigGammaBestSLim.Fill(SigValues[flag][2],SigValues[flag][0]);
    scatterPlotGammaMeanBestSLim[flag].Fill(SigValues[flag][1],SigValues[flag][2]);
    }else{
    if(MeanLimit[flag]<0.001){
            scatterPlotChiMeanBestMLim.Fill(SigValues[flag][1],bestChiScatter);
            scatterPlotSigMeanBestMLim.Fill(SigValues[flag][1],SigValues[flag][0]);
            scatterPlotChiSigBest.Fill(SigValues[flag][0],bestChiScatter);
            scatterPlotChiGammaBest.Fill(SigValues[flag][2],bestChiScatter);
            scatterPlotSigGammaBest.Fill(SigValues[flag][2],SigValues[flag][0]);
            scatterPlotGammaMeanBestMLim[flag].Fill(SigValues[flag][1],SigValues[flag][2]);
    }else{
    if(GammaLimit[flag]<0.001){
            scatterPlotChiMeanBest.Fill(SigValues[flag][1],bestChiScatter);
            scatterPlotSigMeanBest.Fill(SigValues[flag][1],SigValues[flag][0]);
            scatterPlotChiSigBest.Fill(SigValues[flag][0],bestChiScatter);
            scatterPlotChiGammaBestGLim.Fill(SigValues[flag][2],bestChiScatter);
            scatterPlotSigGammaBestGLim.Fill(SigValues[flag][2],SigValues[flag][0]);
            scatterPlotGammaMeanBestGLim[flag].Fill(SigValues[flag][1],SigValues[flag][2]);
    }
    }}

#endif

#ifdef GOODPLOTS
////////////////////////////////////////////////////////////////////////////////////
//PLOTTING GOOD SIGNAL FIT
if(bestChi>=GOODPLOTS){
//////////////////////////////////////////////////////////////////////////////////////////////////
////SAVING TOYMC DISTRIBUTION
//////////////////////////////////////////////////////////////////////////////////////////////////
	sprintf(bufferstring,"ToyMC Good %d",l);	
	genHist->SetTitle(bufferstring);
	genHist->SetName(bufferstring);
	genHist->Write();
	sprintf(bufferstring,"genHist");	
	genHist->SetTitle(bufferstring);
	genHist->SetName(bufferstring);
//////////////////////////////////////////////////////////////////////////////////////////////////
////SAVING PARAMETERS
//////////////////////////////////////////////////////////////////////////////////////////////////
	//paramFile<<SigValues[BestFit][0]<<endl;
	//paramFile<<SigValues[BestFit][1]<<endl;
	//paramFile<<SigValues[BestFit][2]<<endl;
	//paramFile<<endl;
    }
#endif

//////////////////////////////////////////////////////////////////////////////////////////////////
//CLEANING BKG FITS

NullPdfPtr->clearCurrentFit();
ThreePdfBPtr->clearCurrentFit();
 ++sectionCounter;
}//COMPLETE CYCLE

  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);

#ifdef SCATTERPLOTS

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//CHI & SIG SCATTER PLOTS MEAN BEST/////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//CHI vs SIG
  scatterPlotChiSigBest.SetFillColor(1);
  scatterPlotChiSigBestSLim.SetFillColor(2);
  scatterPlotChiSigBest.Write();
  scatterPlotChiSigBestSLim.Write();
  
  scatterPlotChiSigBestSLim.SetStats(0);
  scatterPlotChiSigBest.Draw("box");
  scatterPlotChiSigBestSLim.Draw("box same");

  sprintf(canvasname,"Chi - Signal Fraction Best");	
  canvas->SetTitle(canvasname);
  canvas->SetName(canvasname);
  /*sprintf(canvasname,"plots/ScatterChiSignalBest.png",k+1);
  canvas->SaveAs(canvasname);*/
  canvas->Write(); 
  canvas->Clear();

//CHI vs MEAN
	scatterPlotChiMeanBest.SetFillColor(1);
   scatterPlotChiMeanBestSLim.SetFillColor(2);
   scatterPlotChiMeanBestMLim.SetFillColor(3);
   scatterPlotChiMeanBestGLim.SetFillColor(4);
   scatterPlotChiMeanBestSLim.Write();
   scatterPlotChiMeanBestMLim.Write();
   scatterPlotChiMeanBestGLim.Write();
   scatterPlotChiMeanBest.Write();
   
   scatterPlotChiMeanBestSLim.SetStats(0);
   scatterPlotChiMeanBestMLim.SetStats(0);
   scatterPlotChiMeanBestGLim.SetStats(0);
   scatterPlotChiMeanBest.Draw("box");
   scatterPlotChiMeanBestSLim.Draw("box same");
   scatterPlotChiMeanBestMLim.Draw("box same");
   scatterPlotChiMeanBestGLim.Draw("box same");
   
   sprintf(canvasname,"Chi - Mean Best");	
   canvas->SetTitle(canvasname);
  canvas->SetName(canvasname);
  /*sprintf(canvasname,"plots/ScatterChiMeanBest.png",k+1);
  canvas->SaveAs(canvasname);*/
  canvas->Write(); 
  canvas->Clear();
   
//CHI vs GAMMA
	scatterPlotChiGammaBest.SetFillColor(1);
   scatterPlotChiGammaBestSLim.SetFillColor(2);
   scatterPlotChiGammaBestMLim.SetFillColor(3);
   scatterPlotChiGammaBestGLim.SetFillColor(4);
   scatterPlotChiGammaBest.Write();
   scatterPlotChiGammaBestSLim.Write();
   scatterPlotChiGammaBestMLim.Write();
   scatterPlotChiGammaBestGLim.Write();
   
   scatterPlotChiGammaBestSLim.SetStats(0);
   scatterPlotChiGammaBestMLim.SetStats(0);
   scatterPlotChiGammaBestGLim.SetStats(0);
   scatterPlotChiGammaBest.Draw("box");
   scatterPlotChiGammaBestSLim.Draw("box same");
   scatterPlotChiGammaBestMLim.Draw("box same");
   scatterPlotChiGammaBestGLim.Draw("box same");
   
   sprintf(canvasname,"Chi - Gamma Best");	
   canvas->SetTitle(canvasname);
  canvas->SetName(canvasname);
  /*sprintf(canvasname,"plots/ScatterChiGammaBest.png",k+1);
  canvas->SaveAs(canvasname);*/
  canvas->Write(); 
  canvas->Clear();
   
//SIG vs MEAN
   scatterPlotSigMeanBest.SetFillColor(1);
   scatterPlotSigMeanBestSLim.SetFillColor(2);
   scatterPlotSigMeanBestMLim.SetFillColor(3);
   scatterPlotSigMeanBest.Write();
   scatterPlotSigMeanBestSLim.Write();
   scatterPlotSigMeanBestMLim.Write();
   
   scatterPlotSigMeanBestSLim.SetStats(0);
   scatterPlotSigMeanBestMLim.SetStats(0);
   scatterPlotSigMeanBest.Draw("box");
   scatterPlotSigMeanBestSLim.Draw("box same");
   scatterPlotSigMeanBestMLim.Draw("box same");
   sprintf(canvasname,"Signal Fraction - Mean Best");	
   canvas->SetTitle(canvasname);
  canvas->SetName(canvasname);
  /*sprintf(canvasname,"plots/ScatterSignalMeanBest.png",k+1);
  canvas->SaveAs(canvasname);*/
  canvas->Write(); 
  canvas->Clear();
  
//SIG vs GAMMA
   scatterPlotSigGammaBest.SetFillColor(1);
   scatterPlotSigGammaBestSLim.SetFillColor(2);
   scatterPlotSigGammaBestGLim.SetFillColor(4);
   scatterPlotSigGammaBest.Write();
   scatterPlotSigGammaBestSLim.Write();
   scatterPlotSigGammaBestGLim.Write();
   
   scatterPlotSigGammaBestSLim.SetStats(0);
   scatterPlotSigGammaBestGLim.SetStats(0);
   scatterPlotSigGammaBest.Draw("box");
   scatterPlotSigGammaBestSLim.Draw("box same");
   scatterPlotSigGammaBestGLim.Draw("box same");
   sprintf(canvasname,"Signal Fraction - Gamma Best");
	
   canvas->SetTitle(canvasname);
  canvas->SetName(canvasname);
  /*sprintf(canvasname,"plots/ScatterSignalGammaBest.png");
  canvas->SaveAs(canvasname);*/
  canvas->Write(); 
  canvas->Clear();

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GAMMA MEAN BEST///////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GAMMA MEAN TOTAL BEST
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  
  for(int k=0;k<8;k++){
  scatterPlotGammaMeanBestGLim[k].SetFillColor(4);
  scatterPlotGammaMeanBestMLim[k].SetFillColor(3);
  scatterPlotGammaMeanBestSLim[k].SetFillColor(2);
  scatterPlotGammaMeanBest[k].SetFillColor(1);
  
  scatterPlotGammaMeanBestGLim[k].Write();
  scatterPlotGammaMeanBest[k].Write();
  scatterPlotGammaMeanBestSLim[k].Write();
  scatterPlotGammaMeanBest[k].Write();
  
  scatterPlotGammaMeanBestGLim[k].SetStats(0);
  scatterPlotGammaMeanBestMLim[k].SetStats(0);
  scatterPlotGammaMeanBestSLim[k].SetStats(0);

  scatterPlotGammaMeanBestGLim[k].Draw("box");
  scatterPlotGammaMeanBest[k].Draw("box same");
  scatterPlotGammaMeanBestSLim[k].Draw("box same");
  scatterPlotGammaMeanBest[k].Draw("box same");
  
  grid->Draw();
	
   sprintf(canvasname,"Gamma-Mean Best (%d)",k+1);	
  canvas->SetTitle(canvasname);
  canvas->SetName(canvasname);
  /*sprintf(canvasname,"plots/ScatterGammaMeanBest%d.png",k+1);
  canvas->SaveAs(canvasname);*/
  canvas->Write(); 
  canvas->Clear();
}
  
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GAMMA MEAN GOOD BEST
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  
  for(int k=0;k<8;k++){
	  scatterPlotGammaMeanBest[k].SetFillColor(1);
	  scatterPlotGammaMeanBest[k].Draw("box");
	  scatterPlotGammaMeanBest[k].Write();
	   grid->Draw();
	   sprintf(canvasname,"Gamma-Mean Best Good (%d)",k+1);	
		canvas->SetTitle(canvasname);
		canvas->SetName(canvasname);
	   /*sprintf(canvasname,"plots/ScatterGammaMeanBestGood%d.png",k+1);
	     canvas->SaveAs(canvasname);*/
		 canvas->Write(); canvas->Clear();
	  
  }


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GAMMA MEAN SLIM BEST
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

for(int k=0;k<8;k++){
	  scatterPlotGammaMeanBestSLim[k].SetFillColor(2);
	  scatterPlotGammaMeanBestSLim[k].Draw("box");
	  scatterPlotGammaMeanBestSLim[k].Write();
	   grid->Draw();
	   sprintf(canvasname,"Gamma-Mean Best SLim(%d)",k+1);	
		canvas->SetTitle(canvasname);
		canvas->SetName(canvasname);
	   /*sprintf(canvasname,"plots/ScatterGammaMeanBestSLim%d.png",k+1);
	     canvas->SaveAs(canvasname);*/
		 canvas->Write(); canvas->Clear();
	  
  }
	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GAMMA MEAN MLIM BEST
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
for(int k=0;k<8;k++){
	  scatterPlotGammaMeanBestMLim[k].SetFillColor(2);
	  scatterPlotGammaMeanBestMLim[k].Draw("box");
	  scatterPlotGammaMeanBestMLim[k].Write();
	   grid->Draw();
	   sprintf(canvasname,"Gamma-Mean Best MLim(%d)",k+1);	
		canvas->SetTitle(canvasname);
		canvas->SetName(canvasname);
	   /*sprintf(canvasname,"plots/ScatterGammaMeanBestMLim%d.png",k+1);
	     canvas->SaveAs(canvasname);*/
		 canvas->Write(); canvas->Clear();
	  
  }
	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//GAMMA MEAN GLIM BEST
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

for(int k=0;k<8;k++){
	  scatterPlotGammaMeanBestMLim[k].SetFillColor(4);
	  scatterPlotGammaMeanBestMLim[k].Draw("box");
	  scatterPlotGammaMeanBestMLim[k].Write();
	  grid->Draw();
	  sprintf(canvasname,"Gamma-Mean Best MLim(%d)",k+1);	
	  canvas->SetTitle(canvasname);
	  canvas->SetName(canvasname);
	 /*sprintf(canvasname,"plots/ScatterGammaMeanBestMLim%d.png",k+1);
	     canvas->SaveAs(canvasname);*/
	  canvas->Write(); canvas->Clear();
	  
  }
  
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#endif
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#ifdef STARTINGPOINTS
 //startingPointsNLL->Draw();
 /*sprintf(canvasname,"plots/%d.%d-StartingPointsNll--%dIters-Log.png",Date,Clock,iter);
 canvas->SaveAs(canvasname);*/
 //canvas->Write(); canvas->Clear();
 
 startingPoints->Draw();
 sprintf(canvasname,"plots/%d.%d-StartingPoints--%dIters-Log.png",Date,Clock,iter);
 canvas->SaveAs(canvasname);
 //canvas->Write(); canvas->Clear();
#endif
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//CHI HISTO LOG/////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  canvas->SetLogy(1);
  //GooFile.cd();
  ToyMCDeltaChisquare.SetMarkerStyle(8);
  ToyMCDeltaChisquare.SetMarkerSize(0.4);
  ToyMCDeltaChisquare.Draw();
  ToyMCDeltaChisquare.SetLineColor(kGreen);
  ToyMCDeltaChisquare.Draw("lsame");
  
  sprintf(canvasname,"plots/%d.%d-DeltaChis--%dIters-Log.eps",Date,Clock,iter);
  canvas->SaveAs(canvasname);
  canvas->Clear();
  
  //ToyMCDeltaChisquare.Write();
  /*
  canvas->Divide(4,2);
  for(int h=0;h<8;h++){
	  canvas->cd(h+1);
	  deltaChiStarts[h].SetLineColor(h+1);
	  deltaChiStarts[h].Draw("LTEXT45");
	  deltaChiStarts[h].Write();
  }
  sprintf(canvasname,"plots/%d.%d-DeltaChisStarts--%dIters-Log.eps",Date,Clock,iter);
  canvas->SaveAs(canvasname);
  canvas->Clear();
  
  deltaChiStarts[0].Draw();
	  
  for(int h=1;h<8;h++){
	  deltaChiStarts[h].Draw("same");
  }
  
  sprintf(canvasname,"plots/%d.%d-DeltaChisStartsCompare--%dIters-Log.eps",Date,Clock,iter);
  canvas->SaveAs(canvasname);
  canvas->Clear();
  */

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //sprintf(filename,"txt_files_imp/%d/%d/%d%d%d/%d-ToyMCDeltaChisGoox%d%d%d-%d-%d-%d.txt",Date,iter,gFix,mFix,sNeg,iter,gFix,mFix,sNeg,Date,Clock,toys);
  ofstream timeFile("./Times/time.txt",std::ofstream::app);
  
  cout<<endl;
  cout<<"=================================================================================="<<endl;
  cout<<"Total cycles            "<< cycles <<endl;
  cout<<"Number of fits          "<<nfits<<endl;
  Double_t myCPUc = (stopCPU - startCPU)*10000;
  cout<<"Computation time:       " << (myCPUc / CLOCKS_PER_SEC) << endl ;
  cout<<"=================================================================================="<<endl;
 timeFile<<(myCPUc / CLOCKS_PER_SEC)<<std::endl;
      return 0;
}
